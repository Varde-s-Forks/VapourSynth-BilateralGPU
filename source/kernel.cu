
#include <hip/hip_runtime.h>
#include <iterator>

#define BLOCK_X 16
#define BLOCK_Y 8

hipGraphExec_t get_graphexec(
    float * d_dst, float * d_src, float * h_buffer, 
    int width, int height, int stride, 
    float sigma_spatial, float sigma_color, int radius, 
    bool use_shared_memory);

template <bool use_shared_memory>
__global__ 
__launch_bounds__(BLOCK_X * BLOCK_Y) 
static void bilateral(
    float * __restrict__ dst, const float * __restrict__ src, 
    int width, int height, int stride, 
    float sigma_spatial, float sigma_color, int radius) {

    const int x = threadIdx.x + blockIdx.x * BLOCK_X;
    const int y = threadIdx.y + blockIdx.y * BLOCK_Y;


    float num {};
    float den {};

    if constexpr (use_shared_memory) {
        extern __shared__ float buffer[
            /* (2 * radius + BLOCK_Y) * (2 * radius + BLOCK_X) */];

        for (int cy = threadIdx.y; cy < 2 * radius + BLOCK_Y; cy += BLOCK_Y) {
            int sy = min(max(cy - static_cast<int>(threadIdx.y) - radius + y, 0), height - 1);
            for (int cx = threadIdx.x; cx < 2 * radius + BLOCK_X; cx += BLOCK_X) {
                int sx = min(max(cx - static_cast<int>(threadIdx.x) - radius + x, 0), width - 1);
                buffer[cy * (2 * radius + BLOCK_X) + cx] = src[sy * stride + sx];
            }
        }

        __syncthreads();
        
        if (x >= width || y >= height)
            return;

        const float center = src[y * stride + x];

        for (int cy = -radius; cy <= radius; ++cy) {
            int sy = cy + radius + threadIdx.y;

            for (int cx = -radius; cx <= radius; ++cx) {
                int sx = cx + radius + threadIdx.x;

                float value = buffer[sy * (2 * radius + BLOCK_X) + sx];

                float space = cy * cy + cx * cx;
                float range = (value - center) * (value - center);

                float weight = expf(space * sigma_spatial + range * sigma_color);

                num += weight * value;
                den += weight;
            }
        }
    } else {
        if (x >= width || y >= height)
            return;

        const float center = src[y * stride + x];

        for (int cy = max(y - radius, 0); cy <= min(y + radius, height - 1); ++cy) {
            for (int cx = max(x - radius, 0); cx <= min(x + radius, width - 1); ++cx) {
                const float value = src[cy * stride + cx];

                float space = (y - cy) * (y - cy) + (x - cx) * (x - cx);
                float range = (value - center) * (value - center);

                float weight = expf(space * sigma_spatial + range * sigma_color);

                num += weight * value;
                den += weight;
            }
        }
    }

    dst[y * stride + x] = num / den;
}

hipGraphExec_t get_graphexec(
    float * d_dst, float * d_src, float * h_buffer, 
    int width, int height, int stride, 
    float sigma_spatial, float sigma_color, int radius, 
    bool use_shared_memory
) {

    size_t pitch { stride * sizeof(float) };

    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    hipGraphNode_t n_HtoD;
    {
        hipMemcpy3DParms copy_params {};
        copy_params.srcPtr = make_hipPitchedPtr(
            h_buffer, pitch, width, height);
        copy_params.dstPtr = make_hipPitchedPtr(
            d_src, pitch, width, height);
        copy_params.extent = make_hipExtent(
            width * sizeof(float), height, 1);
        copy_params.kind = hipMemcpyHostToDevice;

        hipGraphAddMemcpyNode(&n_HtoD, graph, nullptr, 0, &copy_params);
    }

    hipGraphNode_t n_kernel;
    {
        hipGraphNode_t dependencies[] { n_HtoD };

        void * kernelArgs[] { 
            &d_dst, &d_src, 
            &width, &height, &stride, 
            &sigma_spatial, &sigma_color, &radius
        };

        hipKernelNodeParams kernel_params {};

        auto sharedMemBytes = static_cast<unsigned int>(
            (2 * radius + BLOCK_Y) * (2 * radius + BLOCK_X) * sizeof(float));
        bool useSharedMem = use_shared_memory && sharedMemBytes < 48 * 1024;

        kernel_params.func = (
            useSharedMem ? 
            reinterpret_cast<void *>(bilateral<true>) : 
            reinterpret_cast<void *>(bilateral<false>)
        );
        kernel_params.blockDim = dim3(BLOCK_X, BLOCK_Y);
        kernel_params.gridDim = dim3(
            (width - 1) / BLOCK_X + 1, 
            (height - 1) / BLOCK_Y + 1
        );
        kernel_params.sharedMemBytes = useSharedMem ? sharedMemBytes : 0;
        kernel_params.kernelParams = kernelArgs;

        hipGraphAddKernelNode(
            &n_kernel, graph, 
            dependencies, std::size(dependencies), 
            &kernel_params);
    }

    hipGraphNode_t n_DtoH;
    {
        hipGraphNode_t dependencies[] { n_kernel };

        hipMemcpy3DParms copy_params {};
        copy_params.srcPtr = make_hipPitchedPtr(
            d_dst, pitch, width, height);
        copy_params.dstPtr = make_hipPitchedPtr(
            h_buffer, pitch, width, height);
        copy_params.extent = make_hipExtent(
            width * sizeof(float), height, 1);
        copy_params.kind = hipMemcpyDeviceToHost;

        hipGraphAddMemcpyNode(
            &n_DtoH, graph, 
            dependencies, std::size(dependencies), 
            &copy_params);
    }

    hipGraphExec_t graphexecp;
    hipGraphInstantiate(&graphexecp, graph, nullptr, nullptr, 0);

    hipGraphDestroy(graph);

    return graphexecp;
}

